/*
  STREAM benchmark implementation in CUDA.

    COPY:       a(i) = b(i)
    SCALE:      a(i) = q*b(i)
    SUM:        a(i) = b(i) + c(i)
    TRIAD:      a(i) = b(i) + q*c(i)

  It measures the memory system on the device.
  The implementation is in double precision.

  Code based on the code developed by John D. McCalpin
  http://www.cs.virginia.edu/stream/FTP/Code/stream.c

  Written by: Massimiliano Fatica, NVIDIA Corporation

  Further modifications by: Ben Cumming, CSCS; Andreas Herten (JSC/FZJ)

  Additional modifications by: Ahmad Rezaii and Brad Chamberlain to
  focus on the Triad kernel for this article
*/

#define NTIMES  10


#include <hip/hip_runtime.h>
#include <string>
#include <vector>

#include <stdio.h>
#include <float.h>
#include <limits.h>
#include <unistd.h>
#include <sys/time.h>

#include <sys/time.h>

# ifndef MIN
# define MIN(x,y) ((x)<(y)?(x):(y))
# endif
# ifndef MAX
# define MAX(x,y) ((x)>(y)?(x):(y))
# endif

typedef double real;

static double   avgtime[4] = {0}, maxtime[4] = {0},
        mintime[4] = {FLT_MAX,FLT_MAX,FLT_MAX,FLT_MAX};


void print_help()
{
    printf(
        "Usage: stream [-s] [-n <elements>] [-b <blocksize>]\n\n"
        "  -s\n"
        "        Print results in SI units (by default IEC units are used)\n\n"
        "  -n <elements>\n"
        "        Put <elements> values in the arrays\n"
        "        (defaults to 1<<26)\n\n"
        "  -b <blocksize>\n"
        "        Use <blocksize> as the number of threads in each block\n"
        "        (defaults to 192)\n"
    );
}

void parse_options(int argc, char** argv, bool& SI, int& N, int& blockSize)
{
    // Default values
    SI = false;
    N = 1<<26;
    blockSize = 192;

    int c;

    while ((c = getopt (argc, argv, "sn:b:h")) != -1)
        switch (c)
        {
            case 's':
                SI = true;
                break;
            case 'n':
                N = std::atoi(optarg);
                break;
            case 'b':
                blockSize = std::atoi(optarg);
                break;
            case 'h':
                print_help();
                std::exit(0);
                break;
            default:
                print_help();
                std::exit(1);
        }
}

/* A gettimeofday routine to give access to the wall
   clock timer on most UNIX-like systems.  */


double mysecond()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp,&tzp);
    return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}


template <typename T>
__global__ void set_array(T * __restrict__ const a, T value, int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        a[idx] = value;
}

template <typename T>
__global__ void STREAM_Triad(T* a, T* b, T* c, T scalar, int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        c[idx] = a[idx] + scalar * b[idx];
}

int main(int argc, char** argv)
{
    real *d_a, *d_b, *d_c;
    int j,k;
    double times[4][NTIMES];
    real scalar;
    std::vector<std::string> label;
    label.push_back("Copy:      ");
    label.push_back("Scale:      ");
    label.push_back("Add:      ");
    label.push_back("Triad:      ");

    // Parse arguments
    bool SI;
    int N, blockSize;
    parse_options(argc, argv, SI, N, blockSize);

    printf(" STREAM Benchmark implementation in CUDA\n");
    printf(" Array size (%s precision) =%7.2f MB\n", sizeof(double)==sizeof(real)?"double":"single", double(N)*double(sizeof(real))/1.e6);

    /* Allocate memory on device */
    hipMalloc((void**)&d_a, sizeof(real)*N);
    hipMalloc((void**)&d_b, sizeof(real)*N);
    hipMalloc((void**)&d_c, sizeof(real)*N);

    /* Compute execution configuration */
    dim3 dimBlock(blockSize);
    dim3 dimGrid(N/dimBlock.x );
    if( N % dimBlock.x != 0 ) dimGrid.x+=1;

    printf(" using %d threads per block, %d blocks\n",dimBlock.x,dimGrid.x);

    if (SI)
        printf(" output in SI units (KB = 1000 B)\n");
    else
        printf(" output in IEC units (KiB = 1024 B)\n");

    /* Initialize memory on the device */
    set_array<real><<<dimGrid,dimBlock>>>(d_a, 2.f, N);
    set_array<real><<<dimGrid,dimBlock>>>(d_b, .5f, N);
    set_array<real><<<dimGrid,dimBlock>>>(d_c, .5f, N);

    /*  --- MAIN LOOP --- repeat test cases NTIMES times --- */

    scalar=3.0f;
    for (k=0; k<NTIMES; k++)
    {
        times[3][k]= mysecond();
        STREAM_Triad<real><<<dimGrid,dimBlock>>>(d_b, d_c, d_a, scalar,  N);
        hipDeviceSynchronize();
        times[3][k]= mysecond() -  times[3][k];
    }

    /*  --- SUMMARY --- */

    for (k=1; k<NTIMES; k++) /* note -- skip first iteration */
    {
        for (j=0; j<4; j++)
        {
            avgtime[j] = avgtime[j] + times[j][k];
            mintime[j] = MIN(mintime[j], times[j][k]);
            maxtime[j] = MAX(maxtime[j], times[j][k]);
        }
    }

    double bytes[4] = {
        2 * sizeof(real) * (double)N,
        2 * sizeof(real) * (double)N,
        3 * sizeof(real) * (double)N,
        3 * sizeof(real) * (double)N
    };

    // Use right units
    const double G = SI ? 1.e9 : static_cast<double>(1<<30);

    printf("\nFunction      Rate %s  Avg time(s)  Min time(s)  Max time(s)\n",
           SI ? "(GB/s) " : "(GiB/s)" );
    printf("-----------------------------------------------------------------\n");
    for (j=3; j<4; j++) {
        avgtime[j] = avgtime[j]/(double)(NTIMES-1);

        printf("%s%11.4f     %11.8f  %11.8f  %11.8f\n", label[j].c_str(),
                bytes[j]/mintime[j] / G,
                avgtime[j],
                mintime[j],
                maxtime[j]);
    }


    /* Free memory on device */
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
